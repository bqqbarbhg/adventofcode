#include <stdio.h>
#include <hip/hip_runtime.h>


struct Target
{
	int2 min;
	int2 max;

	__device__ bool contains(int2 pos) const
	{
		return pos.x >= min.x && pos.x <= max.x
			&& pos.y >= min.y && pos.y <= max.y;
	}

};

__device__ int simulate(Target target, int2 init_vel)
{
	int2 vel = init_vel;
	int2 pos = make_int2(0, 0);
	int max_y = 0;
	while (pos.y >= target.min.y) {
		pos.x += vel.x;
		pos.y += vel.y;
		vel.x += vel.x != 0 ? (vel.x > 0 ? -1 : 1) : 0;
		vel.y -= 1;
		max_y = max(pos.y, max_y);
		if (target.contains(pos)) return max_y;
	}
	return -1;
}

__global__ void find_max_y(Target target, int *p_max_y)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	int max_y = simulate(target, make_int2(x, y));
	atomicMax(p_max_y, max_y);
}

int main(int argc, char **argv)
{
	Target target;
	scanf("target area: x=%d..%d, y=%d..%d", &target.min.x, &target.max.x, &target.min.y, &target.max.y);

	int mx = abs(target.min.x);
	int my = abs(target.min.y);

	int *p_max_y;
	hipMallocManaged(&p_max_y, sizeof(int));
	*p_max_y = 0;

	dim3 block_threads = { 8, 8 };
	dim3 num_blocks = { (mx*4+block_threads.x-1)/block_threads.x, (my*4+block_threads.y-1)/block_threads.y };
	find_max_y<<<num_blocks, block_threads>>>(target, p_max_y);

	int max_y = 0;
	hipMemcpy(&max_y, p_max_y, sizeof(int), hipMemcpyDefault);
	hipFree(p_max_y);

	printf("%d\n", max_y);
	return 0;
}
