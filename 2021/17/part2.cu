#include <stdio.h>
#include <hip/hip_runtime.h>


struct Target
{
	int2 min;
	int2 max;

	__device__ bool contains(int2 pos) const
	{
		return pos.x >= min.x && pos.x <= max.x
			&& pos.y >= min.y && pos.y <= max.y;
	}

};

__device__ bool simulate(Target target, int2 init_vel)
{
	int2 vel = init_vel;
	int2 pos = make_int2(0, 0);
	int max_y = 0;
	while (pos.y >= target.min.y) {
		pos.x += vel.x;
		pos.y += vel.y;
		vel.x += vel.x != 0 ? (vel.x > 0 ? -1 : 1) : 0;
		vel.y -= 1;
		max_y = max(pos.y, max_y);
		if (target.contains(pos)) return true;
	}
	return false;
}

__global__ void find_max_y(Target target, int x0, int y0, int *p_max_y)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	bool hit = simulate(target, make_int2(x0 + x, y0 + y));
	if (hit) {
		atomicAdd(p_max_y, 1);
	}
}

int main(int argc, char **argv)
{
	Target target;
	scanf("target area: x=%d..%d, y=%d..%d", &target.min.x, &target.max.x, &target.min.y, &target.max.y);

	int mx = abs(target.min.x);
	int my = abs(target.min.y);

	int *p_max_y;
	hipMallocManaged(&p_max_y, sizeof(int));
	*p_max_y = 0;

	dim3 block_threads = { 8, 8 };
	dim3 num_blocks = { (mx*4+block_threads.x-1)/block_threads.x, (my*4+block_threads.y-1)/block_threads.y };
	int x0 = mx * -2, y0 = my * -2;
	find_max_y<<<num_blocks, block_threads>>>(target, x0, y0, p_max_y);

	int max_y = 0;
	hipMemcpy(&max_y, p_max_y, sizeof(int), hipMemcpyDefault);
	hipFree(p_max_y);

	printf("%d\n", max_y);
	return 0;
}
